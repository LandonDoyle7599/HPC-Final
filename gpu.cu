#include "hip/hip_runtime.h"
#include "serial.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <vector>
#include <cmath>
#include <chrono>
#include <ctime>

using namespace std;

 // Define a GPU device function to calculateDistance
 __device__ float calculateDistance(Point3D point, Point3D centroid) {
    float dx = point.x - centroid.x;
    float dy = point.y - centroid.y;
    float dz = point.z - centroid.z;
    return sqrt(dx * dx + dy * dy + dz * dz);
}

// Define a GPU kernel to perform k-means clustering
__global__ void kMeansClusteringKernel(Point3D *points, Point3D *centroids, int nPoints, int numCentroids) {
    // Get thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Exit if we are out of bounds
    if (tid >= nPoints) {
        return;
    }
    float minDist = calculateDistance(points[tid], centroids[0]); // setup first point
    int clusterId = 0; // setup first cluster id
    for (int i = 1; i < numCentroids; ++i) {
        float dist = calculateDistance(points[tid], centroids[i]); // calculate distance between point and centroid with GPU function
        if (dist < minDist) {
            minDist = dist;
            clusterId = i;
        }
    }
    // Update cluster id and minimum distance for this point
    points[tid].cluster = clusterId;
    points[tid].minDist = minDist;
}

/**
 * Perform k-means clustering with a GPU
 * @param points - pointer to vector of points
 * @param numEpochs - number of k means iterations
 * @param k - the number of initial centroids
 */
void kMeansClusteringGPU(vector<Point3D> *points, int numEpochs, int numCentroids)
{
  // Initialize centroids
  vector<Point3D> centroids = initializeCentroids(numCentroids, points, true);

  // Run k-means clustering over number of numEpochs to converge the centroids
  for (int i = 0; i < numEpochs; ++i)
  {
    // Allocate memory on GPU
    Point3D *d_points;
    Point3D *d_centroids;
    hipMalloc(&d_points, points->size() * sizeof(Point3D));
    hipMalloc(&d_centroids, centroids.size() * sizeof(Point3D));

    // Copy data to GPU
    hipMemcpy(d_points, points->data(), points->size() * sizeof(Point3D), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids.data(), centroids.size() * sizeof(Point3D), hipMemcpyHostToDevice);

    // Run kernel to compute distance from centroid to each point
    int threadsPerBlock = 1024;
    int blocksPerGrid = (int)ceil((float)points->size() / threadsPerBlock);
    // cout << "Blocks per Grid " << blocksPerGrid << endl;
    kMeansClusteringKernel<<<blocksPerGrid, threadsPerBlock>>>(d_points, d_centroids, points->size(), numCentroids);

    // Copy data back to CPU
    hipMemcpy(points->data(), d_points, points->size() * sizeof(Point3D), hipMemcpyDeviceToHost);
    hipMemcpy(centroids.data(), d_centroids, centroids.size() * sizeof(Point3D), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_points);
    hipFree(d_centroids);

    // Update centroids
    updateCentroidData(points, &centroids, numCentroids);
  }
}

void performGPUKMeans(int numEpochs, int numClusters)
{
    // First we use the same readcsv function as in serial.cpp. TODO: Use the parallel version of this to read in the values
    cout << "Reading the csv" << endl;
    vector<Point3D> points = readcsv("song_data.csv");

    cout << "Entering the k means computation" << endl;
    // Time code: https://stackoverflow.com/questions/21856025/getting-an-accurate-execution-time-in-c-micro-seconds
    auto start_time = std::chrono::high_resolution_clock::now();
    kMeansClusteringGPU(&points, numEpochs, numClusters);
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    cout << "\nSTATS: " << endl;
    cout << "Total points " << points.size() << endl;
    cout << "Epochs " << numEpochs << endl;
    cout << "Clusters: " << numClusters << endl;
    cout << "Time: " << duration.count() << endl;
    cout << endl;
    cout << "Saving the output" << endl;
    saveOutputs(&points, "single-gpu-output.csv");
}

// Use this to run the program and compare outputs
int main() {
  // performGPUKMeans(100, 6);
  // performGPUKMeans(200, 6);
  // performGPUKMeans(100, 12);
  performGPUKMeans(200, 12);
  // performGPUKMeans(600, 12);
  // performGPUKMeans(1200, 12);

  // bool res = areFilesEqual("single-gpu-output.csv", "serialOutput.csv", true);
  // std::cout << "Testing: " <<  res << std::endl;
}


