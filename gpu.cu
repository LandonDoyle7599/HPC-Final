#include "hip/hip_runtime.h"
// #include "serial.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <vector>
#include <cmath>
#include <chrono>
#include <ctime>

using namespace std;

 // Define a GPU device function to calculateDistance
 __device__ float calculateDistance(Point3D point, Point3D centroid) {
    float dx = point.x - centroid.x;
    float dy = point.y - centroid.y;
    float dz = point.z - centroid.z;
    return sqrt(dx * dx + dy * dy + dz * dz);
}

// Define a GPU kernel to perform k-means clustering
__global__ void kMeansClusteringKernel(Point3D *points, Point3D *centroids, int nPoints, int numCentroids) {
    // Get thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Exit if we are out of bounds
    if (tid >= nPoints) {
        return;
    }
    float minDist = calculateDistance(points[tid], centroids[0]); // setup first point
    int clusterId = 0; // setup first cluster id
    for (int i = 1; i < numCentroids; ++i) {
        float dist = calculateDistance(points[tid], centroids[i]); // calculate distance between point and centroid with GPU function
        if (dist < minDist) {
            minDist = dist;
            clusterId = i;
        }
    }
    // Update cluster id and minimum distance for this point
    points[tid].cluster = clusterId;
    points[tid].minDist = minDist;
}

/**
 * Perform k-means clustering with a GPU
 * @param points - pointer to vector of points
 * @param numEpochs - number of k means iterations
 * @param k - the number of initial centroids
 */
void kMeansClusteringGPU(vector<Point3D> *points, int numEpochs, vector<Point3D> *centroids)
{
  // Run k-means clustering over number of numEpochs to converge the centroids
  for (int i = 0; i < numEpochs; ++i)
  {
    // Allocate memory on GPU
    Point3D *d_points;
    Point3D *d_centroids;
    hipMalloc(&d_points, points->size() * sizeof(Point3D));
    hipMalloc(&d_centroids, centroids->size() * sizeof(Point3D));

    // Copy data to GPU
    hipMemcpy(d_points, points->data(), points->size() * sizeof(Point3D), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids->data(), centroids->size() * sizeof(Point3D), hipMemcpyHostToDevice);

    // Run kernel to compute distance from centroid to each point
    int threadsPerBlock = 256;
    int blocksPerGrid = (int)ceil((float)points->size() / threadsPerBlock);
    // cout << "Blocks per Grid " << blocksPerGrid << endl;
    kMeansClusteringKernel<<<blocksPerGrid, threadsPerBlock>>>(d_points, d_centroids, points->size(), centroids->size());

    // Copy data back to CPU
    hipMemcpy(points->data(), d_points, points->size() * sizeof(Point3D), hipMemcpyDeviceToHost);
    hipMemcpy(centroids->data(), d_centroids, centroids->size() * sizeof(Point3D), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_points);
    hipFree(d_centroids);

    // Update centroids
    updateCentroidData(points, centroids, centroids->size());
  }
}

void performGPU(int numEpochs, int numCentroids, vector<Point3D> *centroids, vector<Point3D> *points, string filename)
{
    cout << "Entering the k means computation" << endl;
    // Time code: https://stackoverflow.com/questions/21856025/getting-an-accurate-execution-time-in-c-micro-seconds
    auto start_time = std::chrono::high_resolution_clock::now();
    kMeansClusteringGPU(points, numEpochs, centroids);
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    printStats(numEpochs, numCentroids, points, duration.count());
    saveOutputs(points, filename);
}

// Use this to run the program and compare outputs
// int main() {
//   performGPUKMeans(100, 6);
// }


