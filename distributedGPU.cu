#include <hip/hip_runtime.h>

#include <cstdlib>
#include <cmath>
#include <iostream>
using namespace std;


__device__ float calculateDistance(float x1, float y1, float z1, float x2, float y2, float z2){
    float x = abs(x1 - x2);
    float y = abs(y1 - y2);
    float z = abs(z1 - z2);
    return (x * x) + (y * y) + (z * z);
} 

__global__ void calculateKMean(double k_x[], double k_y[], double k_z[], double recv_x[], double recv_y[], double recv_z[], int assign[], int numLocalDataPoints, int numCentroids){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if we are out of bounds
    if (i > numLocalDataPoints){
        return;
    }
    double min_dist = calculateDistance(k_x[0], k_y[0], k_z[0], recv_x[i], recv_y[i], recv_z[i]);
    int clusterID = 0;
    for (int j = 1; j < numCentroids; ++j)
    // Find the closest centroid
    {
        double temp_dist = calculateDistance(k_x[j], k_y[j], k_z[j], recv_x[i], recv_y[i], recv_z[i]);

        if (temp_dist < min_dist)
        {
            min_dist = temp_dist;
            clusterID = j;
        }
    }
    // Update the assignment
    assign[i] = clusterID;
}

extern "C" {
    void launchCalculateKMean(double k_x[], double k_y[], double k_z[], double recv_x[], double recv_y[], double recv_z[], int assign[], int numLocalDataPoints, int numCentroids){
        int threadsPerBlock = 256;
        int blocksPerGrid = (numLocalDataPoints / threadsPerBlock) + 1;
        // Allocate memory on the GPU
        double *d_k_x, *d_k_y, *d_k_z, *d_recv_x, *d_recv_y, *d_recv_z;
        int *d_assign;
        hipMalloc((void **)&d_k_x, numCentroids * sizeof(double));
        hipMalloc((void **)&d_k_y, numCentroids * sizeof(double));
        hipMalloc((void **)&d_k_z, numCentroids * sizeof(double));
        hipMalloc((void **)&d_recv_x, numLocalDataPoints * sizeof(double));
        hipMalloc((void **)&d_recv_y, numLocalDataPoints * sizeof(double));
        hipMalloc((void **)&d_recv_z, numLocalDataPoints * sizeof(double));
        hipMalloc((void **)&d_assign, numLocalDataPoints * sizeof(int));
        // Copy data to the GPU
        hipMemcpy(d_k_x, k_x, numCentroids * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_k_y, k_y, numCentroids * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_k_z, k_z, numCentroids * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_recv_x, recv_x, numLocalDataPoints * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_recv_y, recv_y, numLocalDataPoints * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_recv_z, recv_z, numLocalDataPoints * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_assign, assign, numLocalDataPoints * sizeof(int), hipMemcpyHostToDevice);
        
        // Launch the kernel
        calculateKMean<<<blocksPerGrid, threadsPerBlock>>>(k_x, k_y, k_z, recv_x, recv_y, recv_z, assign, numLocalDataPoints, numCentroids);
        hipDeviceSynchronize();
        // Copy the result back
        hipMemcpy(assign, d_assign, numLocalDataPoints * sizeof(int), hipMemcpyDeviceToHost);
        // Free the memory
        hipFree(d_k_x);
        hipFree(d_k_y);
        hipFree(d_k_z);
        hipFree(d_recv_x);
        hipFree(d_recv_y);
        hipFree(d_recv_z);
        hipFree(d_assign);
    }
}
