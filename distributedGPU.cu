#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ calculateKMean(double k_x[], double k_y[], double k_z[], double recv_x[], double recv_y[], double recv_z[], int assign[], int numLocalDataPoints, int numCentroids){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if we are out of bounds
    if (i > numLocalDataPoints){
        return;
    }
    double min_dist = numeric_limits<double>::max();
    int clusterID = 0;
    for (int j = 0; j < numCentroids; ++j)
    // Find the closest centroid
    {
        double x = abs(recv_x[i] - k_x[j]);
        double y = abs(recv_y[i] - k_y[j]);
        double z = abs(recv_z[i] - k_z[j]);
        double temp_dist = (x * x) + (y * y) + (z * z);

        if (temp_dist < min_dist)
        {
            min_dist = temp_dist;
            clusterID = j;
        }
    }
    // Update the assignment
    assign[i] = clusterID;
}

void launchCalculateKMean(double k_x[], double k_y[], double k_z[], double recv_x[], double recv_y[], double recv_z[], int assign[], int numLocalDataPoints, int numCentroids){
    int blockSize = 256;
    int numBlocks = (numLocalDataPoints + blockSize - 1) / blockSize;
    // Allocate memory on the GPU
    double *d_k_x, *d_k_y, *d_k_z, *d_recv_x, *d_recv_y, *d_recv_z;
    int *d_assign;
    hipMalloc((void **)&d_k_x, numCentroids * sizeof(double));
    hipMalloc((void **)&d_k_y, numCentroids * sizeof(double));
    hipMalloc((void **)&d_k_z, numCentroids * sizeof(double));
    hipMalloc((void **)&d_recv_x, numLocalDataPoints * sizeof(double));
    hipMalloc((void **)&d_recv_y, numLocalDataPoints * sizeof(double));
    hipMalloc((void **)&d_recv_z, numLocalDataPoints * sizeof(double));
    hipMalloc((void **)&d_assign, numLocalDataPoints * sizeof(int));
    // Copy data to the GPU
    hipMemcpy(d_k_x, k_x, numCentroids * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_k_y, k_y, numCentroids * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_k_z, k_z, numCentroids * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_recv_x, recv_x, numLocalDataPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_recv_y, recv_y, numLocalDataPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_recv_z, recv_z, numLocalDataPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_assign, assign, numLocalDataPoints * sizeof(int), hipMemcpyHostToDevice);
    // Launch the kernel
    calculateKMean<<<numBlocks, blockSize>>>(k_x, k_y, k_z, recv_x, recv_y, recv_z, assign, numLocalDataPoints, numCentroids);
    hipDeviceSynchronize();
    // Copy the result back
    hipMemcpy(assign, d_assign, numLocalDataPoints * sizeof(int), hipMemcpyDeviceToHost);
    // Free the memory
    hipFree(d_k_x);
    hipFree(d_k_y);
    hipFree(d_k_z);
    hipFree(d_recv_x);
    hipFree(d_recv_y);
    hipFree(d_recv_z);
    hipFree(d_assign);
}